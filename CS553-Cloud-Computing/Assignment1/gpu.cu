/*Ron Pyka
  CS 553
  Assignment 1
  GPU Benchmark */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

#define BLOCK_SIZE 16

/* Arrays */
volatile float A[1], B[1000], C[1000000], D[10000][10000];
volatile int E[10000][10000];

/* Initialize D and E*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
  for (col = 0; col < 10000; col++) {
    for (row = 0; row < 10000; row++) {
      D[row][col] = (float)rand() / 32768.0;
      E[row][col] = (int)rand() / 32768;
    }
  }

}

__global__ void gpuFlopTest(float *X){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    X[row*10000 + col] = X [row*10000 + col] * 2.2;
}

__global__ void gpuIopTest(int *X){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    X[row*10000 + col] = X [row*10000 + col] + 2;
}

int main(void)
{
    /* Timing variables */
    struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long usecstart, usecstop;

    int sizeA = 1*sizeof(float);
    int sizeB = 1000*sizeof(float);
    int sizeC = 1000000*sizeof(float);
    int sizeD = 10000*10000*sizeof(float);
    int sizeE = 10000*10000*sizeof(int);
	float *X;
	int *Y;

	int K = ceil((float)10000/((float)BLOCK_SIZE));

    dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(K, K);

    printf("Testing memory speed\n\n");
	hipMalloc(&X, sizeA);

    printf("4B write.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<500000;i++){
    	hipMemcpy(X,(void **)A,sizeA,hipMemcpyHostToDevice);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

    printf("4B read.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<500000;i++){
	    hipMemcpy((void **)A,X,sizeA,hipMemcpyDeviceToHost);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

	hipFree(&X);

	hipMalloc(&X, sizeB);

    printf("4KB write.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<500000;i++){
    	hipMemcpy(X,(void **)B,sizeB,hipMemcpyHostToDevice);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

    printf("4KB read.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<500000;i++){
	    hipMemcpy((void **)B,X,sizeB,hipMemcpyDeviceToHost);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

    hipFree(&X);

	hipMalloc(&X, sizeC);

    printf("4MB write.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<50000;i++){
    	hipMemcpy(X,(void **)C,sizeC,hipMemcpyHostToDevice);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

    printf("4MB read.\n");
	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
	for(int i=0;i<50000;i++){
	    hipMemcpy((void **)C,X,sizeC,hipMemcpyDeviceToHost);
	}
	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

	hipFree(&X);

	printf("\n\nTesting Flops and Iops\n\n");

	initialize_inputs();

	printf("Flops.\n");

	hipMalloc(&X, sizeD);
    hipMemcpy(X,(void **)D,sizeD,hipMemcpyHostToDevice);

	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);

    gpuFlopTest<<<grid, threadBlock>>>(X);

	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

	hipFree(&X);

	printf("Iops.\n");

	hipMalloc(&Y, sizeE);
    hipMemcpy(Y,(void **)E,sizeE,hipMemcpyHostToDevice);

	printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);

    gpuIopTest<<<grid, threadBlock>>>(Y);

	gettimeofday(&etstop, &tzdummy);
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

	hipFree(&Y);
}
